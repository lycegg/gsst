

#include <hip/hip_runtime.h>
#include <stdio.h>
//#include <iostream>
//#include <fstream>
#define SIZE_START 0
#define VER_START 4
#define ZER_START 4+8
#define LEN_START 4+8+1
#define FUHAO_START 4+8+1+8

#define CHECK(call)                                   \
do                                                    \
{                                                     \
    const hipError_t error_code = call;              \
    if (error_code != hipSuccess)                    \
    {                                                 \
        printf("CUDA Error:\n");                      \
        printf("    File:       %s\n", __FILE__);     \
        printf("    Line:       %d\n", __LINE__);     \
        printf("    Error code: %d\n", error_code);   \
        printf("    Error text: %s\n",                \
            hipGetErrorString(error_code));          \
        exit(1);                                      \
    }                                                 \
} while (0)



void __global__ decode(unsigned char*reading,unsigned char*writing,unsigned long long* code_table,unsigned char* codelen,int yuanshi){
    const int n = blockDim.x * blockIdx.x + threadIdx.x;
    int start=0;
    int sum=reading[0];
    if(n!=0)start=reading[n*3]+reading[n*3-1]*256+reading[n*3-2]*65536;
    int end=reading[n*3+3]+reading[n*3+2]*256+reading[n*3+1]*65536;
    writing+=yuanshi*n;
    if(n!=0)writing+=reading[3*sum+n];
    reading+=4*sum;
    for(int i=start;i<end;i++){
        if(reading[i]!=255){
            memcpy(writing,&(code_table[reading[i]]),codelen[reading[i]]);
            writing+=codelen[reading[i]];
        }
        else{
            i++;
            *writing=reading[i];
            writing++;
        }
    }
    //printf("from%dto%d\n",start,end);
}
int huanyuan(unsigned char*x){
    return x[0]*65536+x[1]*256+x[2];
}
int main(int argc, char **argv){
    if (argc != 3) 
    {
        printf("usage: %s infile outfile\n", argv[0]);
        exit(1);
    }
    FILE *fin=fopen(argv[1],"rb"),*fout=fopen(argv[2],"wb");
    int rawsize;
    unsigned char*reading_buf_cpu,*writing_buf_cpu,*reading_buf_gpu,*writing_buf_gpu;
    reading_buf_cpu=(unsigned char*)malloc(65536*256+65536);

    fread(reading_buf_cpu,1,4+17,fin);
    int pos=4+17;
    rawsize=huanyuan(reading_buf_cpu)*256+reading_buf_cpu[3];
    //int sum=reading_buf_cpu[4+17+3];
    //fread(reading_buf_cpu+4+17+3+1,1,sum*4-1,fin);
    unsigned char codelen[256];
    unsigned long long code[256];
    unsigned char*codecount=reading_buf_cpu+4+8+1;
    int i = 0;
    //int pos=4+17+3+1+sum*4-1;
    int sum_fuhao=0;
    //int textsize=huanyuan(&reading_buf_cpu[4+17+3+1+3*(sum-1)+1])+reading_buf_cpu[pos-1];
    //int yuanshi_meikuai=huanyuan(&reading_buf_cpu[4+17]);
    for(int k=0;k<8;k++) {
        sum_fuhao+=codecount[k]*(k+1);
    }
    fread(reading_buf_cpu+pos,1,sum_fuhao,fin);
    for(int kk=0;kk<8;kk++) {
        int k=(kk+1)%8;
        for(int j=0;j<codecount[k];j++){
            codelen[i]=k+1;
            memcpy(code+i,reading_buf_cpu+pos,k+1);
            pos+=k+1;
            i++;
        }
    }
    fread(reading_buf_cpu+pos,1,4,fin);
    int yuanshi_meikuai=huanyuan(reading_buf_cpu+pos);
    int sum=reading_buf_cpu[pos+3];
    int gpu_start_pos=pos+3;
    pos+=4;
    fread(reading_buf_cpu+pos,1,sum*4-1,fin);
    int textsize=huanyuan(&reading_buf_cpu[pos+3*(sum-1)+1])+reading_buf_cpu[pos+sum*4-1-1];
    pos+=sum*4-1;
    //assert()
    fread(reading_buf_cpu+pos,1,textsize,fin);
    //reading_buf_gpu,*writing_buf_gpu
    hipMalloc((void **)&reading_buf_gpu, 65536*256+65536);
    hipMalloc((void **)&writing_buf_gpu, rawsize+65536);
    hipMemcpy(reading_buf_gpu,reading_buf_cpu+gpu_start_pos,1+4*sum-1+textsize,hipMemcpyHostToDevice);
    //CHECK(cudaGetLastError());
    //CHECK(cudaDeviceSynchronize());
    unsigned char* codelen_gpu;
    unsigned long long* code_gpu;
    hipMalloc((void **)&codelen_gpu, 256);
    hipMalloc((void **)&code_gpu, 256*8);
    hipMemcpy(code_gpu,code,256*8,hipMemcpyHostToDevice);
    hipMemcpy(codelen_gpu,codelen,256,hipMemcpyHostToDevice);
    CHECK(hipGetLastError());
    CHECK(hipDeviceSynchronize());
    decode<<<1,sum>>>(reading_buf_gpu,writing_buf_gpu,code_gpu,codelen_gpu,yuanshi_meikuai);
    printf("sum=%d\n",sum);
    CHECK(hipGetLastError());
    CHECK(hipDeviceSynchronize());

    writing_buf_cpu=(unsigned char*)malloc(rawsize+65536);
    hipMemcpy(writing_buf_cpu,writing_buf_gpu,rawsize,hipMemcpyDeviceToHost);
    fwrite(writing_buf_cpu,1,rawsize,fout);
    fclose(fin);
    fclose(fout);
}