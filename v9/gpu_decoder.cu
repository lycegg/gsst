

#include <hip/hip_runtime.h>
#include <stdio.h>
#include <sys/stat.h>
#include <string>
#include <time.h>
#include <vector>
//#include <iostream>
//#include <fstream>
//#define SIZE_START 0
//#define VER_START 4
///#define ZER_START 4+8
//#define LEN_START 4+8+1
//#define FUHAO_START 4+8+1+8
using namespace std;
#define CHECK(call)                                   \
do                                                    \
{                                                     \
    const hipError_t error_code = call;              \
    if (error_code != hipSuccess)                    \
    {                                                 \
        printf("CUDA Error:\n");                      \
        printf("    File:       %s\n", __FILE__);     \
        printf("    Line:       %d\n", __LINE__);     \
        printf("    Error code: %d\n", error_code);   \
        printf("    Error text: %s\n",                \
            hipGetErrorString(error_code));          \
        exit(1);                                      \
    }                                                 \
} while (0)
typedef pair<unsigned char*,int> ppi;

int is_end_of_file(FILE *file) {
    struct stat file_stat;
    
    // 获取文件的状态信息
    if (fstat(fileno(file), &file_stat) != 0) {
        perror("fstat error");
        return -1;
    }

    long current_pos = ftell(file);  // 获取当前文件指针位置
    return current_pos == file_stat.st_size;  // 如果文件指针位置等于文件大小，表示已到达文件末尾
}


void __global__ decode(unsigned char**preading,unsigned char**pwriting,unsigned long long** pcode_table,unsigned char** pcodelen,int *p_RawPerThread){
    unsigned char* reading=preading[blockIdx.x];
    unsigned char*writing=pwriting[blockIdx.x];
    unsigned long long*code_table= pcode_table[blockIdx.x];
    unsigned char*codelen= pcodelen[blockIdx.x];
    int RawPerThread=p_RawPerThread[blockIdx.x];
    const int n = threadIdx.x;
    int start=0;
    int sum=reading[0];
    if(n!=0)start=reading[n*3]+reading[n*3-1]*256+reading[n*3-2]*65536;
    int end=reading[n*3+3]+reading[n*3+2]*256+reading[n*3+1]*65536;
    writing+=RawPerThread*n;
    if(n!=0)writing+=reading[3*sum+n];
    reading+=4*sum;
    for(int i=start;i<end;i++){
        if(reading[i]!=255){
            memcpy(writing,&(code_table[reading[i]]),codelen[reading[i]]);
            writing+=codelen[reading[i]];
        }
        else{
            i++;
            *writing=reading[i];
            writing++;
        }
    }
    //printf("from%dto%d\n",start,end);
}
int DESERIALIZE(unsigned char*x){
    return x[0]*65536+x[1]*256+x[2];
}
vector<ppi> readed,to_write;
FILE *fin,*fout;
long whole_count=0;
int file_stat_size;
void myread(){
    if(whole_count>=file_stat_size)return;
    
    int size;
    fread(&size,4,1,fin);whole_count+=4;
    unsigned char*buf=(unsigned char*)malloc(size);
    whole_count+=size;
    fread(buf,1,size,fin);
    readed.push_back(ppi(buf,size));
}
int main(int argc, char **argv){
    if (argc != 6) 
    {
        printf("usage: %s infile outfile debug parallelismS parallelismT\nparallelismS:total SM num\nparallelismT:thread num per SM\n", argv[0]);
        exit(1);
    }
    int debug_=std::stoi(argv[3]);
    int parallelism=std::stoi(argv[4]);
    int threads_perSM=std::stoi(argv[5]);
    fin=fopen(argv[1],"rb");fout=fopen(argv[2],"wb");
    unsigned char*reading_buf_cpu[parallelism],*whole_reading_buf_gpu[2][parallelism],*whole_writing_buf_gpu[2][parallelism];//,*writing_buf_cpu[parallelism]
    unsigned char **whole_preading_buf_gpu[2],**whole_pwriting_buf_gpu[2];
    unsigned char* whole_codelen_gpu[2][parallelism];
    unsigned char** whole_pcodelen_gpu[2];
    unsigned long long* whole_code_gpu[2][parallelism];
    unsigned long long** whole_pcode_gpu[2];
    int*whole_p_RawPerThreadGpu[2];
    for(int i=0;i<parallelism;i++){
        //reading_buf_cpu[i]=(unsigned char*)malloc(65536*256+65536);
        //writing_buf_cpu[i]=(unsigned char*)malloc(65536*256*8+65536);
        hipMalloc((void **)&whole_reading_buf_gpu[0][i], 65536*256+65536);
        hipMalloc((void **)&whole_writing_buf_gpu[0][i], 4*1024*1025);
        hipMalloc((void **)&whole_codelen_gpu[0][i], 256);
        hipMalloc((void **)&whole_code_gpu[0][i], 256*8);
        hipMalloc((void **)&whole_reading_buf_gpu[1][i], 65536*256+65536);
        hipMalloc((void **)&whole_writing_buf_gpu[1][i], 4*1024*1025);
        hipMalloc((void **)&whole_codelen_gpu[1][i], 256);
        hipMalloc((void **)&whole_code_gpu[1][i], 256*8);
    }
        hipMalloc((void **)&whole_p_RawPerThreadGpu[0], parallelism*sizeof(int));
        hipMalloc((void **)&whole_preading_buf_gpu[0], parallelism*sizeof(int*));
        hipMalloc((void **)&whole_pwriting_buf_gpu[0], parallelism*sizeof(int*));
        hipMalloc((void **)&whole_pcodelen_gpu[0], parallelism*sizeof(int*));
        hipMalloc((void **)&whole_pcode_gpu[0], parallelism*sizeof(int*));
        hipMalloc((void **)&whole_p_RawPerThreadGpu[1], parallelism*sizeof(int));
        hipMalloc((void **)&whole_preading_buf_gpu[1], parallelism*sizeof(int*));
        hipMalloc((void **)&whole_pwriting_buf_gpu[1], parallelism*sizeof(int*));
        hipMalloc((void **)&whole_pcodelen_gpu[1], parallelism*sizeof(int*));
        hipMalloc((void **)&whole_pcode_gpu[1], parallelism*sizeof(int*));
    long sum_time=0;
    //long sum_time_mem=0;
    //cudaStream_t cuda_streams[2];
    struct stat file_stat;
    if (fstat(fileno(fin), &file_stat) != 0) {
        perror("fstat error");
        return -1;
    }
    file_stat_size=file_stat.st_size;
    //unsigned char* whole_reading_file=(unsigned char*)malloc(file_stat_size);
    //fread(whole_reading_file,1,file_stat_size,fin);
    int last_cnt=0;
    int rawsize[2][parallelism];
    int streamcnt=0;
    long time0=clock();
    long iotime=0;
        long time1=clock();
    for(int i=0;i<parallelism;i++)
        myread();
        iotime+=clock()-time1;
    int writeidx=0;
    for(int process_idx=0;process_idx<readed.size();process_idx+=parallelism){
        printf("round_start\n");
        int cur_s=streamcnt&1;
        auto reading_buf_gpu=whole_reading_buf_gpu[cur_s];
        auto writing_buf_gpu=whole_writing_buf_gpu[cur_s];
        auto codelen_gpu=whole_codelen_gpu[cur_s];
        auto code_gpu=whole_code_gpu[cur_s];
        auto p_RawPerThreadGpu=whole_p_RawPerThreadGpu[cur_s];
        auto preading_buf_gpu=whole_preading_buf_gpu[cur_s];
        auto pwriting_buf_gpu=whole_pwriting_buf_gpu[cur_s];
        auto pcodelen_gpu=whole_pcodelen_gpu[cur_s];
        auto pcode_gpu=whole_pcode_gpu[cur_s];
        int RawPerThread[parallelism];
        int cnt=0;
        int numthreads[parallelism];
        printf("clock=%lld\n",clock());
        for(int iii=0;iii<parallelism;iii++){
            //whole_count+=4;
            reading_buf_cpu[iii]=readed[process_idx+iii].first;
            //writing_buf_cpu[iii]=(unsigned char*)malloc(4*1024*1025);
            //long current_pos = 
            //if(debug_)
            //printf("cur_pos:%lld\n",ftell(fin));
            if(process_idx+iii>=readed.size())break;
            cnt++;
            //fread(reading_buf_cpu[iii],1,4+17,fin);
            //whole_count+=4+17;
            int pos=4+17;
            rawsize[cur_s][iii]=DESERIALIZE(reading_buf_cpu[iii])*256+reading_buf_cpu[iii][3];
            //if(debug_)
            //printf("rawsize=%d\n",readed[process_idx+iii].second);
            //int sum=reading_buf_cpu[4+17+3];
            //fread(reading_buf_cpu+4+17+3+1,1,sum*4-1,fin);
            unsigned char codelen[parallelism][256];
            unsigned long long code[parallelism][256];
            unsigned char*codecount=reading_buf_cpu[iii]+4+8+1;
            int i = 0;
            //int pos=4+17+3+1+sum*4-1;
            int sum_fuhao=0;
            //int textsize=DESERIALIZE(&reading_buf_cpu[4+17+3+1+3*(sum-1)+1])+reading_buf_cpu[pos-1];
            //int RawPerThread=DESERIALIZE(&reading_buf_cpu[4+17]);
            for(int k=0;k<8;k++) {
                sum_fuhao+=codecount[k]*(k+1);
            }
            //fread(reading_buf_cpu[iii]+pos,1,sum_fuhao,fin);
            //whole_count+=sum_fuhao;
            for(int kk=0;kk<8;kk++) {
                int k=(kk+1)%8;
                for(int j=0;j<codecount[k];j++){
                    codelen[iii][i]=k+1;
                    memcpy(code[iii]+i,reading_buf_cpu[iii]+pos,k+1);
                    pos+=k+1;
                    i++;
                }
            }
            //fread(reading_buf_cpu[iii]+pos,1,4,fin);
            //whole_count+=4;
            RawPerThread[iii]=DESERIALIZE(reading_buf_cpu[iii]+pos);
            numthreads[iii]=reading_buf_cpu[iii][pos+3];
            int gpu_start_pos=pos+3;
            pos+=4;
            //fread(reading_buf_cpu[iii]+pos,1,numthreads[iii]*4-1,fin);
            //whole_count+=numthreads[iii]*4-1;
            int textsize=DESERIALIZE(&reading_buf_cpu[iii][pos+3*(numthreads[iii]-1)]);//+reading_buf_cpu[iii][pos+numthreads[iii]*4-1-1]
            
            if(debug_)
            printf("yasuosize=%d\n",textsize);
            pos+=numthreads[iii]*4-1;
            //assert()
            //fread(reading_buf_cpu[iii]+pos,1,textsize,fin);
            //whole_count+=textsize;
            //reading_buf_gpu,*writing_buf_gpu
            //CHECK(cudaGetLastError());
            //CHECK(cudaDeviceSynchronize());
            //long time1=clock();
            hipMemcpyAsync(reading_buf_gpu[iii],reading_buf_cpu[iii]+gpu_start_pos,1+4*numthreads[iii]-1+textsize,hipMemcpyHostToDevice);
            //CHECK(cudaGetLastError());
            //CHECK(cudaDeviceSynchronize());
            hipMemcpyAsync(code_gpu[iii],code[iii],256*8,hipMemcpyHostToDevice);
            hipMemcpyAsync(codelen_gpu[iii],codelen[iii],256,hipMemcpyHostToDevice);
            //sum_time_mem+=clock()-time1;
            //CHECK(cudaGetLastError());
            //CHECK(cudaDeviceSynchronize());
            if(debug_)
            printf("numthreads[iii]=%d\n",numthreads[iii]);

        }
        printf("clock=%lld\n",clock());
        //long time1=clock();
        hipMemcpyAsync(p_RawPerThreadGpu,RawPerThread,parallelism*sizeof(int),hipMemcpyHostToDevice);
        //for(int iii=0;iii<cnt;iii++){
        hipMemcpyAsync(preading_buf_gpu,reading_buf_gpu, parallelism*sizeof(int*),hipMemcpyHostToDevice);
        hipMemcpyAsync(pwriting_buf_gpu,writing_buf_gpu, parallelism*sizeof(int*),hipMemcpyHostToDevice);
        hipMemcpyAsync(pcodelen_gpu,codelen_gpu, parallelism*sizeof(int*),hipMemcpyHostToDevice);
        hipMemcpyAsync(pcode_gpu,code_gpu, parallelism*sizeof(int*),hipMemcpyHostToDevice);
        printf("clock=%lld\n",clock());
        //long time0=clock();
        time1=clock();
        for(int iter=0;iter<parallelism;iter++){
            myread();
            if(process_idx>=3*parallelism){
                free(readed[writeidx].first);
                fwrite(to_write[writeidx].first,1,to_write[writeidx].second,fout);
                free(to_write[writeidx].first);
                writeidx++;
            }
        }
        iotime+=clock()-time1;
        printf("clock=%lld\n",clock());
        hipDeviceSynchronize();
        printf("clock=%lld\n",clock());
        decode<<<cnt,threads_perSM>>>(preading_buf_gpu,pwriting_buf_gpu,pcode_gpu,pcodelen_gpu,p_RawPerThreadGpu);
        printf("clock=%lld\n",clock());
        //if(debug_)
        //    printf("finish_time=%lld\n",clock());
        //sum_time+=clock()-time0;
        //if(streamcnt)
        for(int iii=0;iii<last_cnt;iii++){
            auto tmp_buf=(unsigned char*)malloc(4*1024*1025);
            hipMemcpyAsync(tmp_buf,whole_writing_buf_gpu[1-cur_s][iii],rawsize[1-cur_s][iii],hipMemcpyDeviceToHost);
            to_write.push_back(ppi(tmp_buf,rawsize[1-cur_s][iii]));
        }
        printf("clock=%lld\n",clock());
        last_cnt=cnt;
        //CHECK(cudaGetLastError());
        //CHECK(cudaDeviceSynchronize());
        //sum_time_mem+=clock()-time1;
        //for(int iii=0;iii<cnt;iii++){
            //fwrite(,1,fout);

        //}
        streamcnt++;
    }
    int cur_s2=streamcnt&1;
    for(int iii=0;iii<last_cnt;iii++){
        auto tmp_buf=(unsigned char*)malloc(4*1024*1025);
        hipMemcpyAsync(tmp_buf,whole_writing_buf_gpu[1-cur_s2][iii],rawsize[1-cur_s2][iii],hipMemcpyDeviceToHost);
        to_write.push_back(ppi(tmp_buf,rawsize[1-cur_s2][iii]));
    }
    CHECK(hipGetLastError());
    CHECK(hipDeviceSynchronize());
    sum_time=clock()-time0;
    time1=clock();
    for(int iiii=writeidx;iiii<to_write.size();iiii++){
        fwrite(to_write[iiii].first,1,to_write[iiii].second,fout);
        free(to_write[iiii].first);
    }
    iotime+=clock()-time1;
    printf("sum_time=%lld,iotime=%lld\n",sum_time,iotime);
    //printf("sum_time_including_mem=%lld\n",sum_time_mem);
    fclose(fout);
    fclose(fin);
    /*for(int i=0;i<parallelism;i++){
        free(reading_buf_cpu[i]);
        free(writing_buf_cpu[i]);
        cudaFree((void **)&reading_buf_gpu[i]);
        cudaFree((void **)&writing_buf_gpu[i]);
        cudaFree((void **)&codelen_gpu[i]);
        cudaFree((void **)&code_gpu[i]);
    }
        cudaFree((void **)&p_RawPerThreadGpu);*/
    return 0;
}
/*
*/