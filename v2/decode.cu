

#include <hip/hip_runtime.h>
#include <stdio.h>
#include <sys/stat.h>
#include <string>
#include <time.h>
//#include <iostream>
//#include <fstream>
#define SIZE_START 0
#define VER_START 4
#define ZER_START 4+8
#define LEN_START 4+8+1
#define FUHAO_START 4+8+1+8

#define CHECK(call)                                   \
do                                                    \
{                                                     \
    const hipError_t error_code = call;              \
    if (error_code != hipSuccess)                    \
    {                                                 \
        printf("CUDA Error:\n");                      \
        printf("    File:       %s\n", __FILE__);     \
        printf("    Line:       %d\n", __LINE__);     \
        printf("    Error code: %d\n", error_code);   \
        printf("    Error text: %s\n",                \
            hipGetErrorString(error_code));          \
        exit(1);                                      \
    }                                                 \
} while (0)


int is_end_of_file(FILE *file) {
    struct stat file_stat;
    
    // 获取文件的状态信息
    if (fstat(fileno(file), &file_stat) != 0) {
        perror("fstat error");
        return -1;
    }

    long current_pos = ftell(file);  // 获取当前文件指针位置
    return current_pos == file_stat.st_size;  // 如果文件指针位置等于文件大小，表示已到达文件末尾
}


void __global__ decode(unsigned char*reading,unsigned char*writing,unsigned long long* code_table,unsigned char* codelen,int yuanshi){
    const int n = blockDim.x * blockIdx.x + threadIdx.x;
    int start=0;
    int sum=reading[0];
    if(n!=0)start=reading[n*3]+reading[n*3-1]*256+reading[n*3-2]*65536;
    int end=reading[n*3+3]+reading[n*3+2]*256+reading[n*3+1]*65536;
    writing+=yuanshi*n;
    if(n!=0)writing+=reading[3*sum+n];
    reading+=4*sum;
    for(int i=start;i<end;i++){
        if(reading[i]!=255){
            memcpy(writing,&(code_table[reading[i]]),codelen[reading[i]]);
            writing+=codelen[reading[i]];
        }
        else{
            i++;
            *writing=reading[i];
            writing++;
        }
    }
    //printf("from%dto%d\n",start,end);
}
int huanyuan(unsigned char*x){
    return x[0]*65536+x[1]*256+x[2];
}
int main(int argc, char **argv){
    if (argc != 5) 
    {
        printf("usage: %s infile outfile parallelism debug\n parallelism:total SM num\n", argv[0]);
        exit(1);
    }
    int parallelism=std::stoi(argv[3]);
    int debug_=std::stoi(argv[4]);
    int threads_perSM=128;
    FILE *fin=fopen(argv[1],"rb"),*fout=fopen(argv[2],"wb");
    unsigned char*reading_buf_cpu[parallelism],*writing_buf_cpu[parallelism],*reading_buf_gpu[parallelism],*writing_buf_gpu[parallelism];
    unsigned char* codelen_gpu[parallelism];
    unsigned long long* code_gpu[parallelism];
    for(int i=0;i<parallelism;i++){
        reading_buf_cpu[i]=(unsigned char*)malloc(65536*256+65536);
        writing_buf_cpu[i]=(unsigned char*)malloc(65536*256*8+65536);
        hipMalloc((void **)&reading_buf_gpu[i], 65536*256+65536);
        hipMalloc((void **)&writing_buf_gpu[i], 65536*256*8+65536);
        hipMalloc((void **)&codelen_gpu[i], 256);
        hipMalloc((void **)&code_gpu[i], 256*8);
    }
    long sum_time=0;
    while(!is_end_of_file(fin)){
        int rawsize[parallelism];
        int yuanshi_meikuai[parallelism];
        int cnt=0;
        int numthreads[parallelism];
        for(int iii=0;iii<parallelism;iii++){
            //long current_pos = 
            if(debug_)
            printf("cur_pos:%lld\n",ftell(fin));
            if(is_end_of_file(fin))break;
            cnt++;
            fread(reading_buf_cpu[iii],1,4+17,fin);
            int pos=4+17;
            rawsize[iii]=huanyuan(reading_buf_cpu[iii])*256+reading_buf_cpu[iii][3];
            if(debug_)
            printf("rawsize=%d\n",rawsize[iii]);
            //int sum=reading_buf_cpu[4+17+3];
            //fread(reading_buf_cpu+4+17+3+1,1,sum*4-1,fin);
            unsigned char codelen[256];
            unsigned long long code[256];
            unsigned char*codecount=reading_buf_cpu[iii]+4+8+1;
            int i = 0;
            //int pos=4+17+3+1+sum*4-1;
            int sum_fuhao=0;
            //int textsize=huanyuan(&reading_buf_cpu[4+17+3+1+3*(sum-1)+1])+reading_buf_cpu[pos-1];
            //int yuanshi_meikuai=huanyuan(&reading_buf_cpu[4+17]);
            for(int k=0;k<8;k++) {
                sum_fuhao+=codecount[k]*(k+1);
            }
            fread(reading_buf_cpu[iii]+pos,1,sum_fuhao,fin);
            for(int kk=0;kk<8;kk++) {
                int k=(kk+1)%8;
                for(int j=0;j<codecount[k];j++){
                    codelen[i]=k+1;
                    memcpy(code+i,reading_buf_cpu[iii]+pos,k+1);
                    pos+=k+1;
                    i++;
                }
            }
            fread(reading_buf_cpu[iii]+pos,1,4,fin);
            yuanshi_meikuai[iii]=huanyuan(reading_buf_cpu[iii]+pos);
            numthreads[iii]=reading_buf_cpu[iii][pos+3];
            int gpu_start_pos=pos+3;
            pos+=4;
            fread(reading_buf_cpu[iii]+pos,1,numthreads[iii]*4-1,fin);
            int textsize=huanyuan(&reading_buf_cpu[iii][pos+3*(numthreads[iii]-1)]);//+reading_buf_cpu[iii][pos+numthreads[iii]*4-1-1]
            
            if(debug_)
            printf("yasuosize=%d\n",textsize);
            pos+=numthreads[iii]*4-1;
            //assert()
            fread(reading_buf_cpu[iii]+pos,1,textsize,fin);
            //reading_buf_gpu,*writing_buf_gpu
            hipMemcpy(reading_buf_gpu[iii],reading_buf_cpu[iii]+gpu_start_pos,1+4*numthreads[iii]-1+textsize,hipMemcpyHostToDevice);
            //CHECK(cudaGetLastError());
            //CHECK(cudaDeviceSynchronize());
            hipMemcpy(code_gpu[iii],code,256*8,hipMemcpyHostToDevice);
            hipMemcpy(codelen_gpu[iii],codelen,256,hipMemcpyHostToDevice);
            //CHECK(cudaGetLastError());
            //CHECK(cudaDeviceSynchronize());
            if(debug_)
            printf("numthreads[iii]=%d\n",numthreads[iii]);

        }
        CHECK(hipGetLastError());
        CHECK(hipDeviceSynchronize());
        long time0=clock();
        for(int iii=0;iii<cnt;iii++){

            decode<<<1,numthreads[iii]>>>(reading_buf_gpu[iii],writing_buf_gpu[iii],code_gpu[iii],codelen_gpu[iii],yuanshi_meikuai[iii]);
            if(debug_)
                printf("time=%lld\n",clock());
        }
        CHECK(hipGetLastError());
        CHECK(hipDeviceSynchronize());
            if(debug_)
                printf("finish_time=%lld\n",clock());
        sum_time+=clock()-time0;
        for(int iii=0;iii<cnt;iii++){
            hipMemcpy(writing_buf_cpu[iii],writing_buf_gpu[iii],rawsize[iii],hipMemcpyDeviceToHost);
            fwrite(writing_buf_cpu[iii],1,rawsize[iii],fout);
        }
    }
    printf("sum_time=%lld\n",sum_time);
    fclose(fin);
    fclose(fout);
    for(int i=0;i<parallelism;i++){
        free(reading_buf_cpu[i]);
        free(writing_buf_cpu[i]);
        hipFree((void **)&reading_buf_gpu[i]);
        hipFree((void **)&writing_buf_gpu[i]);
        hipFree((void **)&codelen_gpu[i]);
        hipFree((void **)&code_gpu[i]);
    }
    return 0;
}